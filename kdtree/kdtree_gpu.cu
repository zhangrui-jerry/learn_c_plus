#include "hip/hip_runtime.h"
#include "kdtree_gpu.h"
#include <hip/hip_runtime.h>

__global__ void kernel(float *data_src, int src_size, 
    float *data_tar, int tar_size, float *res)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // res += i * tar_size;
    auto x = data_src[i * 2];
    auto y = data_src[i * 2 + 1];
    for (int j = 0; j < tar_size; j++)
    {
        auto x1 = data_tar[j * 2];
        auto y1 = data_tar[j * 2 + 1];
        res[i * tar_size + j] = (x - x1) * (x - x1) + (y - y1) * (y - y1);
    }
}


std::vector<PointT> kdsearch_gpu(const std::vector<PointT>& src, const std::vector<PointT>& tar)
{
    float *res;
    float *src_v;
    float *tar_v;

    hipMallocManaged(&res, src.size() * tar.size() * sizeof(float));
    hipMallocManaged(&src_v, src.size() * 2 * sizeof(float));
    hipMallocManaged(&tar_v, tar.size() * 2 * sizeof(float));

    for (int i = 0; i < src.size(); i++)
    {
        src_v[i * 2] = src[i][0];
        src_v[i * 2 + 1] = src[i][1];
    }

    for (int i = 0; i < tar.size(); i++)
    {
        tar_v[i * 2] = tar[i][0];
        tar_v[i * 2 + 1] = tar[i][1];
    }


    dim3 threads = dim3(256);
    dim3 blocks = dim3(src.size() / threads.x);
    kernel<<<blocks, threads>>>(src_v, src.size(), tar_v, tar.size(), res);
    hipDeviceSynchronize();

    std::vector<PointT> result(src.size(), {0.0, 0.0});
    for (int i = 0; i < src.size(); i++)
    {
        int best_idx = 0;
        float min_dist = res[0];
        for (int j = 0; j < tar.size(); j++)
        {
            auto dist = res[i * tar.size() + j];
            if (dist < min_dist)
            {
                min_dist = dist;
                best_idx = j;
            }
        }
        result[i] = tar[best_idx];
    }

    hipFree(src_v);
    hipFree(tar_v);
    hipFree(res);
    return result;
}

template<typename scalar_t>
__global__ void sided_distance_forward_cuda_kernel(
    int b, int n, const scalar_t * xyz,
    int m, const scalar_t * xyz2,
    scalar_t * result, int64_t * result_i) {
  const int batch=512;
  __shared__ scalar_t buf[batch*3];

  for (int i = blockIdx.x; i<b; i += gridDim.x){
    for (int k2 = 0; k2 < m; k2 += batch) {

      int end_k =  min(m, k2 + batch) - k2;

      for (int j = threadIdx.x; j < end_k * 3; j += blockDim.x) {
        buf[j]=xyz2[(i*m+k2)*3+j];
      }

      __syncthreads();

      for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
        scalar_t x1 = xyz[(i * n + j) * 3 + 0];
        scalar_t y1 = xyz[(i * n + j) * 3 + 1];
        scalar_t z1 = xyz[(i * n + j) * 3 + 2];

        int64_t best_i = 0;
        scalar_t best = 0;
        int end_ka = end_k - (end_k & 3);

        if (end_ka == batch){
          for (int k = 0; k < batch; k += 4) {
            {
            scalar_t x2 = buf[k * 3 + 0] - x1;
            scalar_t y2 = buf[k * 3 + 1] - y1;
            scalar_t z2 = buf[k * 3 + 2]- z1;
            scalar_t d = x2 * x2 + y2 * y2 + z2 * z2;

            if (k == 0 || d < best) {
              best = d;
              best_i = k + k2;
            }
            }

            {
            scalar_t x2 = buf[k * 3 + 3] - x1;
            scalar_t y2 = buf[k * 3 + 4] - y1;
            scalar_t z2 = buf[k * 3 + 5] - z1;
            scalar_t d = x2 * x2 + y2 * y2 + z2 * z2;

            if (d < best){
              best = d;
              best_i = k + k2 + 1;
            }
            }

            {
            scalar_t x2 = buf[k * 3 + 6]- x1;
            scalar_t y2 = buf[k * 3 + 7] - y1;
            scalar_t z2 = buf[k * 3 + 8] - z1;
            scalar_t d = x2 * x2 + y2 * y2 + z2 * z2;

            if (d < best) {
              best = d;
              best_i = k + k2 + 2;
            }
            }

            {
            scalar_t x2 = buf[k * 3 + 9] - x1;
            scalar_t y2 = buf[k * 3 + 10]-y1;
            scalar_t z2 = buf[k*3 + 11] - z1;
            scalar_t d = x2 * x2 + y2 * y2 + z2 * z2;

            if (d < best) {
              best = d;
              best_i = k + k2 + 3;
            }
            }
          }
        } else {
          for (int k = 0; k < end_ka; k += 4) {
            {
              scalar_t x2 = buf[k * 3 + 0] - x1;
              scalar_t y2 = buf[k * 3 + 1] - y1;
              scalar_t z2 = buf[k * 3 + 2] - z1;
              scalar_t d = x2 * x2 + y2 * y2 + z2 * z2;

              if (k == 0 || d < best) {
                best = d;
                best_i = k + k2;
              }
            }

            {
              scalar_t x2 = buf[k * 3 + 3] - x1;
              scalar_t y2 = buf[k * 3 + 4] - y1;
              scalar_t z2 = buf[k * 3 + 5] - z1;
              scalar_t d = x2 * x2 + y2 * y2 + z2 * z2;

              if (d < best) {
                best = d;
                best_i = k + k2 + 1;
              }
            }

            {
              scalar_t x2 = buf[k * 3 + 6] - x1;
              scalar_t y2 = buf[k * 3 + 7] - y1;
              scalar_t z2 = buf[k * 3 + 8] - z1;
              scalar_t d= x2 * x2 + y2 * y2 + z2 * z2;

              if (d < best) {
                best = d;
                best_i = k + k2 + 2;
              }
            }

            {
              scalar_t x2 = buf[k * 3 + 9] - x1;
              scalar_t y2 = buf[k * 3 + 10] - y1;
              scalar_t z2 = buf[k * 3 + 11] - z1;
              scalar_t d = x2 * x2 + y2 * y2 + z2 * z2;

              if (d < best) {
                best = d;
                best_i = k + k2 + 3;
              }
            }
          }
        }
        for (int k = end_ka; k < end_k; k++) {
          scalar_t x2 = buf[k * 3 + 0] - x1;
          scalar_t y2 = buf[k * 3 + 1] - y1;
          scalar_t z2 = buf[k * 3 + 2] - z1;
          scalar_t d = x2 * x2 + y2 * y2 + z2 * z2;

          if (k == 0 || d < best) {
            best = d;
            best_i = k+k2;
          }
        }

        if (k2 == 0 || result[(i * n + j)] > best) {
          result[(i * n + j)] = best;
          result_i[(i * n + j)] = best_i;
        }
      }
      __syncthreads();
    }
  }
}

template<typename scalar_t>
__global__ void sided_distance_forward_cuda_kernel_2d(
    int b, int n, const scalar_t * xyz,
    int m, const scalar_t * xyz2,
    scalar_t * result, int64_t * result_i) {
  const int batch=512;
  constexpr int dim = 2;
  __shared__ scalar_t buf[batch*dim];

  for (int i = blockIdx.x; i<b; i += gridDim.x){
    for (int k2 = 0; k2 < m; k2 += batch) {

      int end_k =  min(m, k2 + batch) - k2;

      for (int j = threadIdx.x; j < end_k * dim; j += blockDim.x) {
        buf[j]=xyz2[(i*m+k2)*dim+j];
      }

      __syncthreads();

      for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
        scalar_t x1 = xyz[(i * n + j) * dim + 0];
        scalar_t y1 = xyz[(i * n + j) * dim + 1];

        int64_t best_i = 0;
        scalar_t best = 0;
        int end_ka = end_k - (end_k & dim);

        best = 10000.0;
        for (int k = 0; k < end_k; k ++) {
            scalar_t x2 = buf[k * dim + 0] - x1;
            scalar_t y2 = buf[k * dim + 1] - y1;
            scalar_t d = x2 * x2 + y2 * y2;

            if (d < best) {
                best = d;
                best_i = k + k2;
            }
        }

        if (k2 == 0 || result[(i * n + j)] > best) {
          result[(i * n + j)] = best;
          result_i[(i * n + j)] = best_i;
        }
      }
      __syncthreads();
    }
  }
}

    void SearchGpu::set_param(int src_size, int tar_size)
    {
        src_size_ = src_size;
        tar_size_ = tar_size;
        hipMallocManaged(&res_, src_size_ * tar_size_ * sizeof(float));
        hipMallocManaged(&src_, src_size_ * 2 * sizeof(float));
        hipMallocManaged(&tar_, tar_size_ * 2 * sizeof(float));
    }

    std::vector<PointT> SearchGpu::kdsearch_gpu(const std::vector<PointT>& src, const std::vector<PointT>& tar)
    {
        for (int i = 0; i < src.size(); i++)
        {
            src_[i * 2] = src[i][0];
            src_[i * 2 + 1] = src[i][1];
        }

        for (int i = 0; i < tar.size(); i++)
        {
            tar_[i * 2] = tar[i][0];
            tar_[i * 2 + 1] = tar[i][1];
        }


        dim3 threads = dim3(256);
        dim3 blocks = dim3(src.size() / threads.x);
        kernel<<<blocks, threads>>>(src_, src_size_, tar_, tar_size_, res_);
        hipDeviceSynchronize();

        std::vector<PointT> result(src.size(), {0.0, 0.0});
        for (int i = 0; i < src.size(); i++)
        {
            int best_idx = 0;
            float min_dist = res_[0];
            for (int j = 0; j < tar_size_; j++)
            {
                auto dist = res_[i * tar_size_ + j];
                if (dist < min_dist)
                {
                    min_dist = dist;
                    best_idx = j;
                }
            }
            result[i] = tar[best_idx];
        }
        return result;
    }

    void SearchGpu::release()
    {
        hipFree(src_);
        hipFree(tar_);
        hipFree(res_);
    }


    void SearchGpu1::set_param(int src_size, int tar_size)
    {
        src_size_ = src_size;
        tar_size_ = tar_size;
        hipMallocManaged(&dist_, src_size_ * sizeof(float));
        hipMallocManaged(&idx_, src_size_ * sizeof(int64_t));
        hipMallocManaged(&src_, src_size_ * 3 * sizeof(float));
        hipMallocManaged(&tar_, tar_size_ * 3 * sizeof(float));
    }

    std::vector<PointT> SearchGpu1::kdsearch_gpu(const std::vector<PointT>& src, const std::vector<PointT>& tar)
    {
        for (int i = 0; i < src.size(); i++)
        {
            src_[i * 3] = src[i][0];
            src_[i * 3 + 1] = src[i][1];
            src_[i * 3 + 2] = 0.0;
        }

        for (int i = 0; i < tar.size(); i++)
        {
            tar_[i * 3] = tar[i][0];
            tar_[i * 3 + 1] = tar[i][1];
            tar_[i * 3 + 2] = 0.0;
        }


        dim3 threads = dim3(256);
        dim3 blocks = dim3(src.size() / threads.x);
        int b = 1;
        // sided_distance_forward_cuda_kernel<float><<<blocks, threads>>>(b, src_size_, src_, tar_size_, tar_, dist_, idx_);
        sided_distance_forward_cuda_kernel<float><<<dim3(32, 16, 1), 512, 0>>>(b, src_size_, src_, tar_size_, tar_, dist_, idx_);
        hipDeviceSynchronize();

        std::vector<PointT> result(src.size(), {0.0, 0.0});
        for (int i = 0; i < src_size_; i++)
        {
            result[i] = tar[idx_[i]];
        }
        return result;
    }

    void SearchGpu1::release()
    {
        hipFree(src_);
        hipFree(tar_);
        hipFree(idx_);
        hipFree(dist_);
    }

    void SearchGpu2::set_param(int src_size, int tar_size)
    {
        src_size_ = src_size;
        tar_size_ = tar_size;
        hipMallocManaged(&dist_, src_size_ * sizeof(float));
        hipMallocManaged(&idx_, src_size_ * sizeof(int64_t));
        hipMallocManaged(&src_, src_size_ * 2 * sizeof(float));
        hipMallocManaged(&tar_, tar_size_ * 2 * sizeof(float));
    }

    std::vector<PointT> SearchGpu2::kdsearch_gpu(const std::vector<PointT>& src, const std::vector<PointT>& tar)
    {
        for (int i = 0; i < src.size(); i++)
        {
            src_[i * 2] = src[i][0];
            src_[i * 2 + 1] = src[i][1];
        }

        for (int i = 0; i < tar.size(); i++)
        {
            tar_[i * 2] = tar[i][0];
            tar_[i * 2 + 1] = tar[i][1];
        }

        int b = 1;
        sided_distance_forward_cuda_kernel_2d<float><<<dim3(32, 16, 1), 512, 0>>>(b, src_size_, src_, tar_size_, tar_, dist_, idx_);
        hipDeviceSynchronize();

        std::vector<PointT> result(src.size(), {0.0, 0.0});
        for (int i = 0; i < src_size_; i++)
        {
            result[i] = tar[idx_[i]];
        }
        return result;
    }

    void SearchGpu2::release()
    {
        hipFree(src_);
        hipFree(tar_);
        hipFree(idx_);
        hipFree(dist_);
    }