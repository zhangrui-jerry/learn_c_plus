#include "hip/hip_runtime.h"
#include <icp.hpp>
#include <data_type/data_type.hpp>
#include <TimeTest/TimeTest.hpp>
#include <kdtree_gpu.h>

namespace icp
{
    template <typename T>
    void generate_cloud(std::shared_ptr<T> cloud, const int &size = 4096 * 8)
    {
        cloud->resize(size);
        auto randd = []
        { return static_cast<double>(rand() % 1024 / 1024.0); };
        for (int i = 0; i < size; i++)
            (*cloud)[i] = {randd(), randd()};
    }

    template <typename Func>
    __global__ void parallel_for(int n, Func func)
    {
        for (int i = threadIdx.x + blockIdx.y * blockDim.x; i < n; i += blockDim.x * gridDim.y)
            func(i);
    }

    template <typename PointType>
    __global__ void search_nn(const PointType *src, const PointType *tar,
                              int src_size, int tar_size, int *indexs, double *dists)
    {
        const int batch = 256;
        __shared__ PointType buf[batch];

        for (int k = 0; k < tar_size; k += batch)
        {
            int batch_size = min(tar_size - k, batch);
            for (int i = threadIdx.x; i < batch_size; i += blockDim.x)
            {
                buf[i] = tar[k + i];
            }

            __syncthreads();

            for (int i = threadIdx.x + blockIdx.y * blockDim.x; i < src_size; i += blockDim.x * gridDim.y)
            {
                const auto &q = src[i];
                int best_index = 0;
                double min_dist = 10000.0;
                for (int j = 0; j < batch_size; j++)
                {
                    const auto &p = buf[j];
                    double dist = (p - q).squaredNorm();
                    if (dist < min_dist)
                    {
                        min_dist = dist;
                        best_index = j + k;
                    }
                }

                if (k == 0 || min_dist < dists[i])
                {
                    dists[i] = min_dist;
                    indexs[i] = best_index;
                }
            }

            __syncthreads();
        }
    }

    int test()
    {
        std::shared_ptr<CloudGPU> src = std::make_shared<CloudGPU>();
        std::shared_ptr<CloudGPU> tar = std::make_shared<CloudGPU>();
        std::shared_ptr<Cloud> src_cpu = std::make_shared<Cloud>();
        std::shared_ptr<Cloud> tar_cpu = std::make_shared<Cloud>();
        std::shared_ptr<Indexs> indexs = std::make_shared<Indexs>();
        std::shared_ptr<Dists> dists = std::make_shared<Dists>();

        INIT_TIME();
        START_TIME("generate cloud");
        generate_cloud(src);
        generate_cloud(tar);
        END_TIME("generate cloud");

        START_TIME("generate cpu cloud");
        generate_cloud(src_cpu);
        generate_cloud(tar_cpu);
        END_TIME("generate cpu cloud");

        START_TIME("create sg");
        SearchGpu sg;
        sg.set_param(src_cpu->size(), tar_cpu->size());
        END_TIME("create sg");

        START_TIME("SG SEARCH");
        auto result = sg.kdsearch_gpu(*(src_cpu), *(tar_cpu));
        END_TIME("SG SEARCH");

        indexs->resize(src->size());
        dists->resize(src->size());
        int b = 1;
        START_TIME("gpu");
        // sided_distance_forward_cuda_kernel_2d<<<dim3(32, 16, 1), 512, 0>>>
        //     ((int)src->size(), src->data(), (int)tar->size(), tar->data(), dists->data(), indexs->data());
        search_nn<<<dim3(32, 16, 1), 512, 0>>>(src->data(), tar->data(), (int)src->size(),
                                               (int)tar->size(), indexs->data(), dists->data());

        hipDeviceSynchronize();
        END_TIME("gpu");

        START_TIME("cpu");
        int count = 0;
        int count1 = 0;
        for (int i = 0; i < src->size(); i++)
        {
            double min_dist = std::numeric_limits<double>::max();
            int best_index = 0;
            for (int j = 0; j < tar->size(); j++)
            {
                double dist = (src->at(i) - tar->at(j)).norm();
                if (dist < min_dist)
                {
                    min_dist = dist;
                    best_index = j;
                }
            }
            if (best_index == indexs->at(i))
                count++;
            if ((result[i] - tar->at(best_index)).norm() < 0.01)
                count1++;
        }
        END_TIME("cpu");
        PRINTF_TIME();
        printf("count %d %d\n", count, count1);
        return 0;
    }
}
