#include "hip/hip_runtime.h"
#include "kdtree_gpu.h"
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void sided_distance_forward_cuda_kernel_2d(
    int b, int n, const scalar_t *xyz,
    int m, const scalar_t *xyz2,
    scalar_t *result, int64_t *result_i)
{
  const int batch = 512;
  constexpr int dim = 2;
  __shared__ scalar_t buf[batch * dim];

  for (int i = blockIdx.x; i < b; i += gridDim.x)
  {
    for (int k2 = 0; k2 < m; k2 += batch)
    {
    
      int end_k = min(m, k2 + batch) - k2;

      for (int j = threadIdx.x; j < end_k * dim; j += blockDim.x)
      {
        buf[j] = xyz2[(i * m + k2) * dim + j];
      }

      __syncthreads();

      for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y)
      {
        scalar_t x1 = xyz[(i * n + j) * dim + 0];
        scalar_t y1 = xyz[(i * n + j) * dim + 1];

        int64_t best_i = 0;
        scalar_t best = 0;
        int end_ka = end_k - (end_k & dim);

        best = 10000.0;
        for (int k = 0; k < end_k; k++)
        {
          scalar_t x2 = buf[k * dim + 0] - x1;
          scalar_t y2 = buf[k * dim + 1] - y1;
          scalar_t d = x2 * x2 + y2 * y2;

          if (d < best)
          {
            best = d;
            best_i = k + k2;
          }
        }

        if (k2 == 0 || result[(i * n + j)] > best)
        {
          result[(i * n + j)] = best;
          result_i[(i * n + j)] = best_i;
        }
      }
      __syncthreads();
    }
  }
}

void SearchGpu::set_param(int src_size, int tar_size)
{
  src_size_ = src_size;
  tar_size_ = tar_size;
  hipMallocManaged(&dist_, src_size_ * sizeof(float));
  hipMallocManaged(&idx_, src_size_ * sizeof(int64_t));
  hipMallocManaged(&src_, src_size_ * 2 * sizeof(float));
  hipMallocManaged(&tar_, tar_size_ * 2 * sizeof(float));
}

std::vector<int> SearchGpu::kdsearch_gpu(const std::vector<PointT> &src, const std::vector<PointT> &tar)
{
  for (int i = 0; i < src.size(); i++)
  {
    src_[i * 2] = src[i][0];
    src_[i * 2 + 1] = src[i][1];
  }

  for (int i = 0; i < tar.size(); i++)
  {
    tar_[i * 2] = tar[i][0];
    tar_[i * 2 + 1] = tar[i][1];
  }

  int b = 1;
  sided_distance_forward_cuda_kernel_2d<float><<<dim3(32, 16, 1), 512, 0>>>(b, src_size_, src_, tar_size_, tar_, dist_, idx_);
  hipDeviceSynchronize();

  std::vector<int> result(src.size());
  for (int i = 0; i < src_size_; i++)
  {
    result[i] = idx_[i];
  }
  return result;
}

void SearchGpu::release()
{
  hipFree(src_);
  hipFree(tar_);
  hipFree(idx_);
  hipFree(dist_);
}