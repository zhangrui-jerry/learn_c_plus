#include "hip/hip_runtime.h"
#include <icp.hpp>
#include <data_type/data_type.hpp>

namespace icp
{
    template<typename T>
    void generate_cloud(std::shared_ptr<T> cloud, const int& size = 1000)
    {
        cloud->resize(size);
        auto randd = []{return static_cast<double>(rand() % 1024 / 1024.0);};
        for (int i = 0; i < size;  i++)
            (*cloud)[i] = {randd(), randd()};
    }

template<typename PointT, typename scalar_t>
__global__ void sided_distance_forward_cuda_kernel_2d(
    int b, int n, const PointT* xyz,
    int m, const PointT* xyz2,
    double* result, int64_t* result_i) 
{
//     const int batch=256;
//     __shared__ PointT buf[batch];

//   {
//     for (int k2 = 0; k2 < m; k2 += batch) {

//       int end_k =  min(m - k2, batch);
//       for (int j = threadIdx.x; j < end_k; j += blockDim.x) {
//         buf[j]=xyz2[k2+j];
//       }

//       __syncthreads();

//       for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
//         scalar_t x1 = xyz[j][0];
//         scalar_t y1 = xyz[j][1];

//         int64_t best_i = 0;
//         scalar_t best = 0;
//         best = 10000.0;
//         for (int k = 0; k < end_k; k ++) {
//             scalar_t x2 = buf[k][0] - x1;
//             scalar_t y2 = buf[k][1] - y1;
//             scalar_t d = x2 * x2 + y2 * y2;

//             if (d < best) {
//                 best = d;
//                 best_i = k + k2;
//             }
//         }

//         if (k2 == 0 || result[j] > best) {
//           result[j] = best;
//           result_i[j] = best_i;
//         }
//       }
//       __syncthreads();
//     }
//   }
}
    int test()
    {
        std::shared_ptr<CloudGPU> src = std::make_shared<CloudGPU>();
        std::shared_ptr<CloudGPU> tar = std::make_shared<CloudGPU>();
        std::shared_ptr<Indexs> indexs = std::make_shared<Indexs>();
        std::shared_ptr<Dists> dists = std::make_shared<Dists>();

        generate_cloud(src);
        generate_cloud(tar);

        indexs->resize(src->size());
        dists->resize(src->size());
        int b = 1;
        sided_distance_forward_cuda_kernel_2d<PointT, double><<<dim3(32, 16, 1), 512, 0>>>
            (b, (int)src->size(), src->data(), (int)tar->size(), tar->data(), dists->data(), indexs->data());

        int count = 0; 
        for (int i = 0; i < src->size(); i++)
        {
            double min_dist = std::numeric_limits<double>::max();
            int best_index = 0;
            for (int j = 0; j < tar->size(); j++)
            {
                double dist = (src->at(i) - tar->at(j)).norm();
                if (dist < min_dist)
                {
                    min_dist = dist;
                    best_index = j;
                }
            }
            if (best_index == indexs->at(i))
                count++;
            // printf("%d %d\n", best_index, indexs->at(i));
        }
        printf("count %d\n", count);
        return 0;
    }
}
